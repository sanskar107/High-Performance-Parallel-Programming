
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void swap(int *M, int mat_size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int N = mat_size;

	if((i < N) && (j < N) && (j%2 == 0) && (j != N - 1))
	{
		int tmp = M[i * mat_size + j];
		M[i * mat_size + j] = M[i*mat_size + j + 1];
		M[i * mat_size + j + 1] = tmp;
	}
	__syncthreads();
}

__global__ void reflect(int *M, int mat_size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int N = mat_size;

	if((i < N) && (j < N) && (i > j))
	{
		M[j*mat_size + i] = M[i*mat_size + j];
	}
	__syncthreads();
}