#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>

__global__ void process_kernel1(const float *, const float *, float *, int);
__global__ void process_kernel2(const float *, float *, int);
__global__ void process_kernel3(const float *, float *, int);

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 16*32*32;
    size_t size = numElements * sizeof(float);
//    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        // if(i < 5) printf("%f\n", h_B[i]);
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    printf("Size: %d\n",size);
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        printf("A failed\n");
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    dim3 X(4, 2, 2);
    dim3 Y(32, 32, 1);

    printf("Running Test 1\n");
    process_kernel1<<<X, Y>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sin(h_A[i]) + cos(h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

  //   for(int i = 0; i < 5; i++)
  //   {
		// printf("%f %f %f \n", h_A[i], h_B[i], h_C[i]);
  //   }
    printf("Test1 PASSED\n");

    float *h_C2 = (float *)malloc(size);
    float *d_C2 = NULL;
    err = hipMalloc((void **)&d_C2, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Running Test 2\n");
    dim3 X2(2, 8, 1);
    dim3 Y2(8, 8, 16);

    process_kernel2<<<X2, Y2>>>(d_C, d_C2, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C2, d_C2, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(log(h_C[i]) - h_C2[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test2 Passed\n");


    float *h_C3 = (float *)malloc(size);
    float *d_C3 = NULL;
    err = hipMalloc((void **)&d_C3, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printf("Running Test 3\n");
    dim3 X3(16, 1, 1);
    dim3 Y3(128, 8, 1);
    
    process_kernel3<<<X3, Y3>>>(d_C2, d_C3, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C3, d_C3, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt(h_C2[i]) - h_C3[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test3 Passed\n");



    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

