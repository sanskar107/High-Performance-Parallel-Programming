
#include <hip/hip_runtime.h>
__global__ void process_kernel1(const float *input1,const  float *input2, float *output, int datasize)
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x+ blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x* blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int i = blockNum * (blockDim.x * blockDim.y * blockDim.z) +threadNum;

	if(i < datasize)
	{
		output[i] = sin(input1[i]) + cos(input2[i]);
	}
}
	

__global__ void process_kernel2(const float *input, float *output, int datasize)
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x+ blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x* blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int i = blockNum * (blockDim.x * blockDim.y * blockDim.z) +threadNum;
	if(i < datasize)
	{
		output[i] = log(input[i]);
	}
}

__global__ void process_kernel3(const float *input, float *output, int datasize)
{
	int blockNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x+ blockIdx.x;
	int threadNum = threadIdx.z * (blockDim.x* blockDim.y) + threadIdx.y * (blockDim.x) + threadIdx.x;
	int i = blockNum * (blockDim.x * blockDim.y * blockDim.z) +threadNum;
	if(i < datasize)
	{
		output[i] = sqrt(input[i]);
	}
}
